

#include "GPUPrimPlugin.h"

void GPUPrimPlugin::gpu(float* a, int Msize, edge* ptrEdges) {
	int numThreads = Msize;//1024;
	int numBlocks = Msize / 1024 + 1;

	hipMemcpy(gpuA, a, Msize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuMatrix, cost, Msize*Msize*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(edgeArray, ptrEdges, Msize*sizeof(edge), hipMemcpyHostToDevice);

	findEdgeOfRow<<<numBlocks, numThreads, Msize*sizeof(edge)>>>(gpuA, Msize, gpuMatrix, edgeArray);

	hipMemcpy(edges, edgeArray, Msize*sizeof(edge), hipMemcpyDeviceToHost);

	hipFree(&gpuA);
	hipFree(&gpuMatrix);
	hipFree(&edgeArray);

}




void GPUPrimPlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;
   parameters[key] = value;
 }
 N = atoi(parameters["N"].c_str());
 visited = (int*) malloc(N*sizeof(int));
 int i;
 for (i = 0; i < N; i++) {
    visited[i] = 0;
 } 
 cost = (int*) malloc(N*N*sizeof(int));
 edges = (edge*) malloc(N*sizeof(edge));
 int M = N * N;
 std::ifstream myinput((std::string(PluginManager::prefix())+parameters["matrix"]).c_str(), std::ios::in);
 for (i = 0; i < M; ++i) {
	int k;
	myinput >> k;
        cost[i] = k;
 }
}

void GPUPrimPlugin::run() {
	float* r = (float*) malloc(N*sizeof(float));
	edge* ptrEdge;
	int k = N;
	int mx;
	int mincost = 0;
		printf("\n------------------%d x %d Matrix----------------------\n", k, k);
		ptrEdge = &edges[0];

		hipMalloc(&gpuA, k*sizeof(float));
        	hipMalloc(&gpuMatrix, ( k* k*sizeof(int)));
	        hipMalloc(&edgeArray, ( k*sizeof(edge)));

		edge* edgeStart = ptrEdge;
	
		visited[0]=1;
	        printf("\n");
	        while(ne<k)
	        {	
			gpu(r,k,ptrEdge);
			ptrEdge = edgeStart;
	
	                for(j=0, mx = 0;j<k;j++)
	                {
	                        if(edges[j].edgeWeight>mx)
	                        {
	                                if(visited[edges[j].ownerVertex]!=0)
	                                {
	                                        mx=edges[j].edgeWeight;
	                                        a=u=j;
	                                        b=v=edges[j].pointVertex;
	                                }
	                        }
	                }	
	                if(visited[u]==0 || visited[v]==0)
	                {
//                             printf("\nEdge %d:(%d %d) cost:%d",ne++,a,b,mx);
				ne++;
	                        mincost+=mx;
	                        visited[b]=1;
	                }
	                cost[a*k+b]=cost[b*k+a]=(cost[a*k+b] * -1);
	        }
	        printf("\nMaximum cost = %d\n",mincost);

}

void GPUPrimPlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
        int i;
        for (i = 0; i < N; ++i){
		outfile << edges[i].ownerVertex;
		outfile << "\t";
		outfile << edges[i].pointVertex;
		outfile << "\t";
		outfile << edges[i].edgeWeight;
		outfile << "\n";//std::setprecision(0) << a[i*N+j];
	}
	

}

PluginProxy<GPUPrimPlugin> GPUPrimPluginProxy = PluginProxy<GPUPrimPlugin>("GPUPrim", PluginManager::getInstance());


